#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <nvblox/integrators/certified_projective_tsdf_integrator.h>
#include <nvblox/integrators/projective_tsdf_integrator.h>

#include "nvblox/geometry/bounding_boxes.h"
#include "nvblox/geometry/bounding_spheres.h"
#include "nvblox/integrators/internal/cuda/impl/projective_integrator_impl.cuh"
#include "nvblox/integrators/internal/integrators_common.h"
#include "nvblox/integrators/weighting_function.h"

namespace nvblox {

struct CertifiedUpdateTsdfVoxelFunctor {
  CertifiedUpdateTsdfVoxelFunctor() {}

  // Vector3f p_voxel_C, float depth, TsdfVoxel* voxel_ptr
  __device__ bool operator()(const float surface_depth_measured_,
                             const float voxel_depth_m, TsdfVoxel* voxel_ptr) {

    // Filter out invalid returns
    float surface_depth_measured = surface_depth_measured_;
    if (surface_depth_measured_ <= min_distance_m_) {
      surface_depth_measured = 7.0; // TODO(rgg): make this a parameter
    }
    // Get the distance between the voxel we're updating the surface.
    // Note that the distance is the projective distance, i.e. the distance
    // along the ray.
    const float voxel_to_surface_distance =
        surface_depth_measured - voxel_depth_m;
    // If we're behind the negative truncation distance, just continue.
    if (voxel_to_surface_distance < -truncation_distance_m_) {
      return false;
    }

    // Read CURRENT voxel values (from global GPU memory)
    const float voxel_distance_current = voxel_ptr->distance;
    // Fuse without using the weights to do en exponential moving average.

    // TODO(rgg): examine whether there is a more efficient way to mark observed
    // voxels? Weight update is needed because it also marks the voxel as
    // observed.

    // Get the weight of this observation from the sensor model.
    const float measurement_weight = weighting_function_(
        surface_depth_measured, voxel_depth_m, truncation_distance_m_);
    // TODO(rgg): remove magic number here
    const float voxel_weight_current = voxel_ptr->weight;
    const float weight =
        measurement_weight +
        0.09;  // 0.001 is the min threshold for observability, 0.1 for "softly"
              //  observed. Adding ~0.06 here seems like a reasonable
              //  compromise, but it'd be better to design a weighting function
              //  that does what we want it to.
    // const float weight =
        // fmin(measurement_weight + voxel_weight_current, max_weight_);
    // Fuse
    float fused_distance = voxel_to_surface_distance;
    // float fused_distance = (voxel_to_surface_distance * measurement_weight +
    //                         voxel_distance_current * voxel_weight_current) /
    //                        (measurement_weight + voxel_weight_current);

    // Clip
    if (fused_distance > 0.0f) {
      fused_distance = fmin(truncation_distance_m_, fused_distance);
    } else {
      fused_distance = fmax(-truncation_distance_m_, fused_distance);
    }
    voxel_ptr->weight = weight;
    // Write NEW voxel values (to global GPU memory)
    voxel_ptr->distance = fused_distance;
    return true;
  }

  float truncation_distance_m_ = 0.2f;
  float min_distance_m_ = 0.10f;  // Minimum distance to consider a return to be valid
  float max_weight_ = 100.0f;

  // TODO(rgg): update this with a new weighting function type
  WeightingFunction weighting_function_ =
      WeightingFunction(WeightingFunctionType::kConstantWeight);
};

CertifiedProjectiveTsdfIntegrator::CertifiedProjectiveTsdfIntegrator()
    : ProjectiveIntegrator<TsdfVoxel>() {
  update_functor_host_ptr_ =
      make_unified<CertifiedUpdateTsdfVoxelFunctor>(MemoryType::kHost);
  checkCudaErrors(hipStreamCreate(&integration_stream_));
}

CertifiedProjectiveTsdfIntegrator::~CertifiedProjectiveTsdfIntegrator() {
  hipStreamSynchronize(integration_stream_);
  checkCudaErrors(hipStreamDestroy(integration_stream_));
}

unified_ptr<CertifiedUpdateTsdfVoxelFunctor>
CertifiedProjectiveTsdfIntegrator::getTsdfUpdateFunctorOnDevice(
    float voxel_size) {
  // Set the update function params
  // NOTE(alex.millane): We do this with every frame integration to avoid
  // bug-prone logic for detecting when params have changed etc.
  update_functor_host_ptr_->max_weight_ = max_weight();
  update_functor_host_ptr_->truncation_distance_m_ =
      get_truncation_distance_m(voxel_size);
  update_functor_host_ptr_->weighting_function_ =
      WeightingFunction(weighting_function_type_);
  // Transfer to the device
  return update_functor_host_ptr_.clone(MemoryType::kDevice);
}

void CertifiedProjectiveTsdfIntegrator::integrateFrame(
    const DepthImage& depth_frame, const Transform& T_L_C, const Camera& camera,
    TsdfLayer* layer, std::vector<Index3D>* updated_blocks) {
  // Get the update functor on the device
  unified_ptr<CertifiedUpdateTsdfVoxelFunctor> update_functor_device_ptr =
      getTsdfUpdateFunctorOnDevice(layer->voxel_size());
  // Integrate
  ProjectiveIntegrator<TsdfVoxel>::integrateFrame(
      depth_frame, T_L_C, camera,
      update_functor_host_ptr_.clone(MemoryType::kDevice).get(), layer,
      updated_blocks);
}

void CertifiedProjectiveTsdfIntegrator::integrateFrame(
    const DepthImage& depth_frame, const Transform& T_L_C, const Lidar& lidar,
    TsdfLayer* layer, std::vector<Index3D>* updated_blocks) {
  // Get the update functor on the device
  unified_ptr<CertifiedUpdateTsdfVoxelFunctor> update_functor_device_ptr =
      getTsdfUpdateFunctorOnDevice(layer->voxel_size());
  // Integrate
  ProjectiveIntegrator<TsdfVoxel>::integrateFrame(
      depth_frame, T_L_C, lidar, update_functor_device_ptr.get(), layer,
      updated_blocks);
}

float CertifiedProjectiveTsdfIntegrator::max_weight() const {
  return max_weight_;
}

void CertifiedProjectiveTsdfIntegrator::max_weight(float max_weight) {
  CHECK_GT(max_weight, 0.0f);
  max_weight_ = max_weight;
}

WeightingFunctionType
CertifiedProjectiveTsdfIntegrator::weighting_function_type() const {
  return weighting_function_type_;
}

void CertifiedProjectiveTsdfIntegrator::weighting_function_type(
    WeightingFunctionType weighting_function_type) {
  weighting_function_type_ = weighting_function_type;
}

float CertifiedProjectiveTsdfIntegrator::marked_unobserved_voxels_distance_m()
    const {
  return marked_unobserved_voxels_distance_m_;
}

void CertifiedProjectiveTsdfIntegrator::marked_unobserved_voxels_distance_m(
    float marked_unobserved_voxels_distance_m) {
  marked_unobserved_voxels_distance_m_ = marked_unobserved_voxels_distance_m;
}

float CertifiedProjectiveTsdfIntegrator::marked_unobserved_voxels_weight()
    const {
  return marked_unobserved_voxels_weight_;
}

void CertifiedProjectiveTsdfIntegrator::marked_unobserved_voxels_weight(
    float marked_unobserved_voxels_weight) {
  marked_unobserved_voxels_weight_ = marked_unobserved_voxels_weight;
}

std::string CertifiedProjectiveTsdfIntegrator::getIntegratorName() const {
  return "certified_tsdf";
}

// Call with:
// - One threadBlock per VoxelBlock
// - 8x8x8 threads per threadBlock
__global__ void setUnobservedVoxelsCertKernel(const TsdfVoxel voxel_value,
                                              TsdfBlock** tsdf_block_ptrs) {
  // Get the voxel addressed by this thread.
  TsdfBlock* tsdf_block = tsdf_block_ptrs[blockIdx.x];
  TsdfVoxel* tsdf_voxel =
      &tsdf_block->voxels[threadIdx.z][threadIdx.y][threadIdx.x];
  // If voxel not observed set it to the constant value input to the kernel.
  // TODO(rgg): examine whether there is a more efficient way to mark observed
  // voxels?
  constexpr float kMinObservedWeight = 0.001;
  if (tsdf_voxel->weight < kMinObservedWeight) {
    *tsdf_voxel = voxel_value;
  }
}

void CertifiedProjectiveTsdfIntegrator::markUnobservedFreeInsideRadius(
    const Vector3f& center, float radius, TsdfLayer* layer,
    std::vector<Index3D>* updated_blocks_ptr) {
  CHECK_NOTNULL(layer);
  CHECK_GT(radius, 0.0f);
  // First get blocks in AABB
  const Vector3f min = center.array() - radius;
  const Vector3f max = center.array() + radius;
  const AxisAlignedBoundingBox aabb(min, max);
  const std::vector<Index3D> blocks_touched_by_aabb =
      getBlockIndicesTouchedByBoundingBox(layer->block_size(), aabb);
  // Narrow to radius
  const std::vector<Index3D> blocks_inside_radius = getBlocksWithinRadius(
      blocks_touched_by_aabb, layer->block_size(), center, radius);
  // Allocate (if they're not already);
  std::for_each(
      blocks_inside_radius.begin(), blocks_inside_radius.end(),
      [layer](const Index3D& idx) { layer->allocateBlockAtIndex(idx); });

  // TsdfBlock pointers to GPU
  const std::vector<TsdfBlock*> block_ptrs_host =
      getBlockPtrsFromIndices(blocks_inside_radius, layer);
  device_vector<TsdfBlock*> block_ptrs_device(block_ptrs_host);

  // The value given to "observed" voxels
  constexpr float kSlightlyObservedVoxelWeight = 0.1;
  const TsdfVoxel slightly_observed_tsdf_voxel{
      .distance = get_truncation_distance_m(layer->voxel_size()),
      .weight = kSlightlyObservedVoxelWeight};

  // Kernel launch
  const int num_thread_blocks = block_ptrs_device.size();
  constexpr int kVoxelsPerSide = TsdfBlock::kVoxelsPerSide;
  const dim3 num_threads_per_block(kVoxelsPerSide, kVoxelsPerSide,
                                   kVoxelsPerSide);
  setUnobservedVoxelsCertKernel<<<num_thread_blocks, num_threads_per_block, 0,
                                  integration_stream_>>>(
      slightly_observed_tsdf_voxel, block_ptrs_device.data());
  hipStreamSynchronize(integration_stream_);
  checkCudaErrors(hipPeekAtLastError());

  // Return blocks affected
  if (updated_blocks_ptr != nullptr) {
    *updated_blocks_ptr = blocks_inside_radius;
  }
}

}  // namespace nvblox
